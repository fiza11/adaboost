#include "hip/hip_runtime.h"
#include<gtest/gtest.h>
#include<string>
#include<adaboost/cuda/core/cuda_data_structures.hpp>
#include<adaboost/cuda/utils/cuda_wrappers.hpp>
#include<adaboost/core/operations.hpp>
#include<adaboost/cuda/core/operations.hpp>
#include<stdexcept>
#include "/usr/include/linux/hip/hip_runtime.h"



__device__ float square_1(float x)
{
    return x*x;
}

__device__ float square_2(float x)
{
    return -x*x;
}

__constant__ adaboost::cuda::core::func_t<float,float> p_func=square_1;

TEST(Cuda, Argmax)
{
    adaboost::utils::cuda::cuda_event_t has_happened;
    adaboost::utils::cuda::cuda_event_create(&has_happened);
    adaboost::cuda::core::VectorGPU<float> vec_i(5);
    vec_i.set(0, -1.0);
    vec_i.set(1, 0);
    vec_i.set(2, 1);
    vec_i.set(3, 2);
    vec_i.set(4, 3);
    unsigned int block_size = 2;
    adaboost::utils::cuda::cuda_event_record(has_happened);
    adaboost::utils::cuda::cuda_event_synchronize(has_happened);
    float result_gpu;

    adaboost::cuda::core::Argmax(p_func, vec_i, result_gpu, block_size);
    adaboost::utils::cuda::cuda_event_record(has_happened);
    adaboost::utils::cuda::cuda_event_synchronize(has_happened);
    EXPECT_EQ(3, result_gpu)<<"The arg max value is at 3.";
}
